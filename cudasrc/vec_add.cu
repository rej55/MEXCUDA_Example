#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"

__global__
void vec_add
(
    const double* src1,
    const double* src2,
    const double k1,
    const double k2,
    double* dst,
    int const N
)
{
    // Calculate index
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < N)
    {
        dst[tid] = k1*src1[tid] + k2*src2[tid];
    }
}

void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, const mxArray *prhs[])
{
    // Define variables
    const mxGPUArray *src1;
    const mxGPUArray *src2;
    double k1;
    double k2;
    mxGPUArray *dst;
    const double *d_src1;
    const double *d_src2;
    double *d_dst;
    int N1, N2;

    // Check the number of arguments
    if ( nrhs != 4 ) {
        mexErrMsgIdAndTxt("MATLAB:vec_add","The number of input arguments must be 4.");
    } 
    if ( nlhs != 1 ) {
        mexErrMsgIdAndTxt("MATLAB:vec_add","The number of output arguments must be 1.");
    } 

    // Initialization
    mxInitGPU();

    // Get data from *prhs[]
    src1 = mxGPUCreateFromMxArray(prhs[0]);
    src2 = mxGPUCreateFromMxArray(prhs[1]);
    k1 = mxGetScalar(prhs[2]);
    k2 = mxGetScalar(prhs[3]);

    // Check the dimension of src vectors
    N1 = (int)(mxGPUGetNumberOfElements(src1));
    N2 = (int)(mxGPUGetNumberOfElements(src2));
    if ( N1 != N2 ) {
        mxGPUDestroyGPUArray(src1);
        mxGPUDestroyGPUArray(src2);
        mexErrMsgIdAndTxt("MATLAB:vec_add","The dimension of input vectors must be same.");
    }

    // Get address of src1 and src2
    d_src1 = (const double*)(mxGPUGetDataReadOnly(src1));
    d_src2 = (const double*)(mxGPUGetDataReadOnly(src2));

    // Allocate memory of the destination variable on device memory
    dst = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(src1),
                            mxGPUGetDimensions(src1),
                            mxGPUGetClassID(src1),
                            mxGPUGetComplexity(src1),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_dst = (double *)(mxGPUGetData(dst));

    // Call kernel function
    dim3 block(N1);
    dim3 grid((N1 + block.x - 1) / block.x);
    vec_add<<<grid, block>>>(d_src1, d_src2, k1, k2, d_dst, N1);

    // Pass dst to plhs[0]
    plhs[0] = mxGPUCreateMxArrayOnGPU(dst);

    // Release memory
    mxGPUDestroyGPUArray(src1);
    mxGPUDestroyGPUArray(src2);
}